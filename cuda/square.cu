﻿
// C

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(
    float* d_out,
    float* d_in) 
  {
  int idx = threadIdx.x;
  float f = d_in[idx];
  d_out[idx] = f * f;
}

int main(int argc, char ** argv) {
  const int ARRAY_SIZE = 64;
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
  
  // generate the input array on the host
  float h_in[ARRAY_SIZE];
  for (int i = 0; i < ARRAY_SIZE; i++) {
    h_in[i] = float(i);
  }
  float h_out[ARRAY_SIZE];
  
  // declare GPU memory pointers
  float * d_in;
  float * d_out;
  
  
  // copy back the result array to the CPU
  hipMemcpy(d_in, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
  
  
  
  return 0;
}